
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <math.h>
#include <assert.h>

#include <unistd.h>




// kernel to make the calculation. 
__global__ void calc(float* a, float* b, float* c, float* x, float* LH)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  // strat with: simple gaussian evaluation. 
  LH[idx] = a[idx]*expf( -1*(x[idx]-b[idx]*b[idx])/(2*c[idx]*c[idx]) );



}



//////////////
int main (int argc, char **argv)
{

// do I have any input args? 
char* name;
if(argc>1)
  {
    name = argv[1];
  }


// how many walkers? They will be evaluated in parallel. 
int nWalkers = 80;


// set up the CPU memory. 
size_t sizeneeded = nWalkers*sizeof(float);
float *h_a = 0;
float *h_b = 0;
float *h_c = 0;
float *h_x = 0;

h_a = (float*) malloc(sizeneeded);
h_b = (float*) malloc(sizeneeded);
h_c = (float*) malloc(sizeneeded);
h_x = (float*) malloc(sizeneeded);

// assign them random numbers. 
for(int i=0;i<nWalkers;i++){
  h_a[i]=0.1;
  h_b[i]=0.5;
  h_c[i]=10.0;
  h_x[i]=i;
}

// assign the GPU memory
float *d_a, *d_b, *d_c, *d_x;
hipMalloc(&d_a, sizeneeded);
hipMalloc(&d_b, sizeneeded);
hipMalloc(&d_c, sizeneeded);
hipMalloc(&d_x, sizeneeded);

// assign the output memory
size_t sizeneeded_out = nWalkers*sizeof(float);
float *h_LH = 0; 
float *d_LH;
h_LH = (float*) malloc(sizeneeded_out);
hipMalloc(&d_LH, sizeneeded_out);

//. copy data over to GPU
hipMemcpy(d_a, h_a, sizeneeded, hipMemcpyHostToDevice);
hipMemcpy(d_b, h_b, sizeneeded, hipMemcpyHostToDevice);
hipMemcpy(d_c, h_c, sizeneeded, hipMemcpyHostToDevice);
hipMemcpy(d_x, h_x, sizeneeded, hipMemcpyHostToDevice);
hipMemcpy(d_LH, h_LH, sizeneeded_out, hipMemcpyHostToDevice);


// set up kernel params. 
// First: 80 walkers, each will eval one gaussian. 
int threadsPerBlock = 1024; // max possible. Don't care much about mem access yet. 
int blocksPerGrid = int(ceil(nWalkers / float(threadsPerBlock)));
    printf(" theads per block: %d and blocks per grid: %d for a total of: %d\n", threadsPerBlock, blocksPerGrid, threadsPerBlock*blocksPerGrid);


// run it! 
calc<<<blocksPerGrid, threadsPerBlock >>> (d_a, d_b, d_c, d_x, d_LH);


// copy the data back off the GPU
hipMemcpy(h_LH, d_LH, sizeneeded_out, hipMemcpyDeviceToHost);

// print it out...
for(int i=0;i<nWalkers;i++){
  printf("LH is: %f \n", h_LH[i]);
}


}